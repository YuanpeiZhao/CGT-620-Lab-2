#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

using namespace std;

__global__ void kernelFun(void) {

	printf("Hello from block %i of %i and thread %i\n",
		blockIdx.x, blockDim.x, threadIdx.x);
}
int main() {

	kernelFun << <16, 16 >> > ();


	int devCnt = 0;
	hipError_t error_id = hipGetDeviceCount(&devCnt);

	int dev, driverVersion = 0, runtimeVersion = 0;
	for (dev = 0; dev < devCnt; ++dev) {
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		cout << "Device " << dev << ": " << deviceProp.name << endl;
		cout << "total global memory: " << deviceProp.totalGlobalMem << endl;
		cout << "clock frequency: " << deviceProp.clockRate << endl;
		cout << "maximum number of threads per block: " << deviceProp.maxThreadsPerBlock << endl;
		cout << "maximum shared memory of a thread block: " << deviceProp.sharedMemPerBlock << endl;
		cout << "number of multiprocessors: " << deviceProp.multiProcessorCount << endl;
		cout << "l2 cache size: " << deviceProp.l2CacheSize << endl << endl;
	}

	
	return 0;
}